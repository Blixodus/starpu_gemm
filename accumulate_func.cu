#include <starpu.h>
#include <starpu_cublas_v2.h>
#include "hipblas.h"
#include <iostream>
#include <sys/syscall.h>

#include "accumulate_func.hpp"

void accumulate_matrix_cpu(void * buffers[], void * cl_args) {
  int m = STARPU_MATRIX_GET_NX(buffers[0]);
  int n = STARPU_MATRIX_GET_NY(buffers[0]);
  int ld_dst = STARPU_MATRIX_GET_LD(buffers[0]);
  float * dst = (float*)STARPU_MATRIX_GET_PTR(buffers[0]);
  int ld_src = STARPU_MATRIX_GET_LD(buffers[1]);
  float * src = (float*)STARPU_MATRIX_GET_PTR(buffers[1]);
  for(int j = 0; j < n; j++) {
    for(int i = 0; i < m; i++) {
      dst[i + ld_dst * j] = dst[i + ld_dst * j] + src[i + ld_src * j];
    }
  }
}

void accumulate_matrix_cuda(void * buffers[], void * cl_args) {
  int m = STARPU_MATRIX_GET_NX(buffers[0]);
  int n = STARPU_MATRIX_GET_NY(buffers[0]);
  int ld_dst = STARPU_MATRIX_GET_LD(buffers[0]);
  float * dst = (float*)STARPU_MATRIX_GET_PTR(buffers[0]);
  int ld_src = STARPU_MATRIX_GET_LD(buffers[1]);
  float * src = (float*)STARPU_MATRIX_GET_PTR(buffers[1]);
  float alpha = 1, beta = 1;
  hipblasStatus_t stat = hipblasSgeam(starpu_cublas_get_local_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &alpha, dst, ld_dst, &beta, src, ld_src, dst, ld_dst);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS GEMM failed\n");
  }
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}

