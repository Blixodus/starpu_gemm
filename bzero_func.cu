#include <starpu.h>
#include <starpu_cublas_v2.h>
#include "hipblas.h"
#include "bzero_func.hpp"

void bzero_matrix_cpu(void * buffers[], void * cl_args) {
  int m = STARPU_MATRIX_GET_NX(buffers[0]);
  int n = STARPU_MATRIX_GET_NY(buffers[0]);
  int ld = STARPU_MATRIX_GET_LD(buffers[0]);
  float * mat = (float*)STARPU_MATRIX_GET_PTR(buffers[0]);
  for(int j = 0; j < n; j++) {
    memset(&mat[ld*j], 0, m*sizeof(float));
  }
}

void bzero_matrix_cuda(void * buffers[], void * cl_args) {
  int m = STARPU_MATRIX_GET_NX(buffers[0]);
  int n = STARPU_MATRIX_GET_NY(buffers[0]);
  int ld = STARPU_MATRIX_GET_LD(buffers[0]);
  float * mat = (float*)STARPU_MATRIX_GET_PTR(buffers[0]);
  float alpha = 0, beta = 0;
  hipblasStatus_t stat = hipblasSgeam(starpu_cublas_get_local_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &alpha, mat, ld, &beta, mat, ld, mat, ld);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS GEMM failed\n");
  }
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}