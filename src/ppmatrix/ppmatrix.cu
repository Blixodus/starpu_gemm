#include "hip/hip_runtime.h"
#include <stdexcept>
#include <fmt/core.h>
#include <chrono>
#include <random>

#include "ppmatrix.hpp"
#include "pputils.hpp"

#include "../util/helper.hpp"
#include "../util/lapackAPI.hpp"

// #include <cutlass/cutlass.h>

template <typename DataType>
void PPMatrix<DataType>::rndFill() {
    std::random_device rd;
    std::mt19937 e2(rd());
    std::uniform_real_distribution<DataType> dist(0, 1000);

    for (u32 i = 0; i < cols; ++i) {
        for (u32 j = 0; j < rows; ++j) {
            ptr[i * ld + j] = dist(e2);
        }
    }
}

template <typename DataType>
void PPMatrix<DataType>::fill(DataType e) {
    for (u32 i = 0; i < cols; ++i) {
        for (u32 j = 0; j < rows; ++j) {
            ptr[i * ld + j] = e;
        }
    }
}

template <typename DataType>
void PPMatrix<DataType>::assertEq(DataType e) {
    for (u32 i = 0; i < cols; ++i) {
        for (u32 j = 0; j < rows; ++j) {
            if (std::abs(ptr[i * ld + j] - e) > 1e-6) {
                fmt::print("Assertion failed at ({}, {}): {} != {}\n", i, j, ptr[i * ld + j], e);
            }
        }
    }
}

PerfRecord ppgemm_f32(
    hipblasHandle_t handle,
    char transA,
    char transB,
    f32 alpha,
    const PPMatrix<f32>& A,
    const PPMatrix<f32>& B,
    f32 beta,
    PPMatrix<f32>& C
) {
    assert(A.rows == C.rows);
    assert(B.cols == C.cols);
    assert(A.cols == B.rows);

    int m = checked_cast<int>((transA == 'N') ? A.rows : A.cols);
    int n = checked_cast<int>((transB == 'N') ? B.cols : B.rows);
    int k = checked_cast<int>((transA == 'N') ? A.cols : A.rows);

    PerfRecord perf;

    // ###########################################
    // STEP 0: preparation
    // ###########################################

    // alloc CPU buffers
    f16 *A_1, *A_2, *A_3, *B_1, *B_2, *B_3, *C_1, *C_2, *C_3;

    HANDLE_ERR(hipHostMalloc(&A_1, A.rows * A.cols * sizeof(f16)));
    HANDLE_ERR(hipHostMalloc(&A_2, A.rows * A.cols * sizeof(f16)));
    HANDLE_ERR(hipHostMalloc(&A_3, A.rows * A.cols * sizeof(f16)));

    HANDLE_ERR(hipHostMalloc(&B_1, B.rows * B.cols * sizeof(f16)));
    HANDLE_ERR(hipHostMalloc(&B_2, B.rows * B.cols * sizeof(f16)));
    HANDLE_ERR(hipHostMalloc(&B_3, B.rows * B.cols * sizeof(f16)));

    HANDLE_ERR(hipHostMalloc(&C_1, C.rows * C.cols * sizeof(f16)));
    HANDLE_ERR(hipHostMalloc(&C_2, C.rows * C.cols * sizeof(f16)));
    HANDLE_ERR(hipHostMalloc(&C_3, C.rows * C.cols * sizeof(f16)));

    // alloc GPU buffers
    f16 *dA_1, *dA_2, *dA_3, *dB_1, *dB_2, *dB_3, *dC_1, *dC_2, *dC_3;

    HANDLE_ERR(hipMalloc(&dA_1, A.rows * A.cols * sizeof(f16)));
    HANDLE_ERR(hipMalloc(&dA_2, A.rows * A.cols * sizeof(f16)));
    HANDLE_ERR(hipMalloc(&dA_3, A.rows * A.cols * sizeof(f16)));

    HANDLE_ERR(hipMalloc(&dB_1, B.rows * B.cols * sizeof(f16)));
    HANDLE_ERR(hipMalloc(&dB_2, B.rows * B.cols * sizeof(f16)));
    HANDLE_ERR(hipMalloc(&dB_3, B.rows * B.cols * sizeof(f16)));

    HANDLE_ERR(hipMalloc(&dC_1, C.rows * C.cols * sizeof(f16)));
    HANDLE_ERR(hipMalloc(&dC_2, C.rows * C.cols * sizeof(f16)));
    HANDLE_ERR(hipMalloc(&dC_3, C.rows * C.cols * sizeof(f16)));

    // use the default stream so that no sync are needed before / after a call to this function
    hipStream_t s0 = 0;

    // register streams
    // streams are created as hipStreamNonBlocking, meaning they do not
    // implicitly sync with the default stream
    hipStream_t s1, s2, s3, s4, s5;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s3, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s4, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s5, hipStreamNonBlocking);

    // register events
    hipEvent_t e0, e1;
    hipEventCreate(&e0);
    hipEventCreate(&e1);

    // ###########################################
    // STEP 1: precision-decompose and send A & B
    // ###########################################

    // decompose A and B on the host (don't start copying yet to get accurate perf numbers)
    de_f32f16(A.ptr, A_1, A_2, A_3, A.rows, A.cols, A.ld);
    de_f32f16(B.ptr, B_1, B_2, B_3, B.rows, B.cols, B.ld);

    auto start = std::chrono::high_resolution_clock::now();

    // copy A_1, B_1, C_1 to GPU
    hipMemcpyAsync(dA_1, A_1, A.rows * A.cols * sizeof(f16), hipMemcpyHostToDevice, s0);
    hipMemcpyAsync(dB_1, B_1, B.rows * B.cols * sizeof(f16), hipMemcpyHostToDevice, s1);
    hipMemcpyAsync(dC_1, C_1, C.rows * C.cols * sizeof(f16), hipMemcpyHostToDevice, s2);

    // copy A_2, B_2, C_2 to GPU
    hipMemcpyAsync(dA_2, A_2, A.rows * A.cols * sizeof(f16), hipMemcpyHostToDevice, s3);
    hipMemcpyAsync(dB_2, B_2, B.rows * B.cols * sizeof(f16), hipMemcpyHostToDevice, s4);
    hipMemcpyAsync(dC_2, C_2, C.rows * C.cols * sizeof(f16), hipMemcpyHostToDevice, s5);

    // ###################################################
    // STEP 2: convert dA_1, dB_1 and dC_1 to fp32
    // ###################################################

    // ###################################################
    // STEP 8 wait for everything to finish
    // ###################################################

    hipStreamSynchronize(s0);
    hipStreamSynchronize(s1);
    hipStreamSynchronize(s2);
    hipStreamSynchronize(s3);
    hipStreamSynchronize(s4);
    hipStreamSynchronize(s5);

    perf.compute = std::chrono::high_resolution_clock::now() - start;

    // ###################################################
    // STEP 9: cleanup
    // ###################################################

    hipStreamDestroy(s0);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);
    hipStreamDestroy(s4);
    hipStreamDestroy(s5);

    hipEventDestroy(e0);
    hipEventDestroy(e1);

    // ###################################################
    // STEP 10: precision-recompose C
    // ###################################################
    re_f32f16(C_1, C_2, C_3, C.ptr, C.rows, C.cols, C.ld);

    hipFree(dA_1);
    hipFree(dA_2);
    hipFree(dA_3);
    hipFree(dB_1);
    hipFree(dB_2);
    hipFree(dB_3);
    hipFree(dC_1);
    hipFree(dC_2);
    hipFree(dC_3);

    hipHostFree(A_1);
    hipHostFree(A_2);
    hipHostFree(A_3);
    hipHostFree(B_1);
    hipHostFree(B_2);
    hipHostFree(B_3);
    hipHostFree(C_1);
    hipHostFree(C_2);
    hipHostFree(C_3);

    return perf;
}

PerfRecord ppgemm_f64(
    hipblasHandle_t handle,
    char transA,
    char transB,
    f64 alpha,
    const PPMatrix<f64>& A,
    const PPMatrix<f64>& B,
    f64 beta,
    PPMatrix<f64>& C
) {
    assert(A.rows == C.rows);
    assert(B.cols == C.cols);
    assert(A.cols == B.rows);

    int m = checked_cast<int>((transA == 'N') ? A.rows : A.cols);
    int n = checked_cast<int>((transB == 'N') ? B.cols : B.rows);
    int k = checked_cast<int>((transA == 'N') ? A.cols : A.rows);

    PerfRecord perf;

    // ###########################################
    // STEP 0: preparation
    // ###########################################

    // alloc CPU buffers
    float *A_h, *A_l, *B_h, *B_l, *C_h, *C_l;
    
    HANDLE_ERR(hipHostMalloc(&A_h, A.rows * A.cols * sizeof(float)));
    HANDLE_ERR(hipHostMalloc(&A_l, A.rows * A.cols * sizeof(float)));

    HANDLE_ERR(hipHostMalloc(&B_h, B.rows * B.cols * sizeof(float)));
    HANDLE_ERR(hipHostMalloc(&B_l, B.rows * B.cols * sizeof(float)));

    HANDLE_ERR(hipHostMalloc(&C_h, C.rows * C.cols * sizeof(float)));
    HANDLE_ERR(hipHostMalloc(&C_l, C.rows * C.cols * sizeof(float)));

    // alloc GPU buffers
    float *dA_h, *dA_l, *dB_h, *dB_l, *dC_h, *dC_l;
    double *dA_dgemm, *dB_dgemm, *dRes_dgemm;

    HANDLE_ERR(hipMalloc(&dA_h, A.rows * A.cols * sizeof(float)));
    HANDLE_ERR(hipMalloc(&dA_l, A.rows * A.cols * sizeof(float)));

    HANDLE_ERR(hipMalloc(&dB_h, B.rows * B.cols * sizeof(float)));
    HANDLE_ERR(hipMalloc(&dB_l, B.rows * B.cols * sizeof(float)));

    HANDLE_ERR(hipMalloc(&dC_h, C.rows * C.cols * sizeof(float)));
    HANDLE_ERR(hipMalloc(&dC_l, C.rows * C.cols * sizeof(float)));

    HANDLE_ERR(hipMalloc(&dA_dgemm, A.rows * A.cols * sizeof(double)));
    HANDLE_ERR(hipMalloc(&dB_dgemm, B.rows * B.cols * sizeof(double)));
    HANDLE_ERR(hipMalloc(&dRes_dgemm, C.rows * C.cols * sizeof(double)));

    // use the default stream so that no sync are needed before / after a call to this function
    hipStream_t s0 = 0;

    // register streams
    // streams are created as hipStreamNonBlocking, meaning they do not
    // implicitly sync with the default stream
    hipStream_t s1, s2, s3;
    HANDLE_ERR(hipStreamCreateWithFlags(&s0, hipStreamNonBlocking));
    HANDLE_ERR(hipStreamCreateWithFlags(&s1, hipStreamNonBlocking));
    HANDLE_ERR(hipStreamCreateWithFlags(&s2, hipStreamNonBlocking));
    HANDLE_ERR(hipStreamCreateWithFlags(&s3, hipStreamNonBlocking));

    // register events
    hipEvent_t e0, e1;
    HANDLE_ERR(hipEventCreate(&e0));
    HANDLE_ERR(hipEventCreate(&e1));

    // ###########################################
    // STEP 1: precision-decompose and send A & B
    // ###########################################

    // No need to use events here, communications from/to the device are serialized

    // decompose A and B on the host (don't start copying yet to get accurate perf numbers)
    de_f64f32(A.ptr, A_h, A_l, A.rows, A.cols, A.ld);
    de_f64f32(B.ptr, B_h, B_l, B.rows, B.cols, B.ld);
    
    auto start = std::chrono::high_resolution_clock::now();

    hello<<<1, 1, 0, s0>>>();
    hello<<<1, 1, 0, s1>>>();
    hello<<<1, 1, 0, s2>>>();
    hello<<<1, 1, 0, s3>>>();

    // copy A_h to GPU
    HANDLE_ERR(hipMemcpyAsync(dA_h, A_h, A.rows * A.cols * sizeof(float), hipMemcpyHostToDevice, s0));

    // copy B_h to GPU
    HANDLE_ERR(hipMemcpyAsync(dB_h, B_h, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice, s2));

    // copy A_l to GPU
    HANDLE_ERR(hipMemcpyAsync(dA_l, A_l, A.rows * A.cols * sizeof(float), hipMemcpyHostToDevice, s1));

    // copy B_l to GPU
    HANDLE_ERR(hipMemcpyAsync(dB_l, B_l, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice, s3));
    HANDLE_ERR(hipEventRecord(e1, s3));

    // ###################################################
    // STEP 2: convert dA_h and dB_h to double
    // ###################################################

    // launch the kernel in blocks of 256 threads
    // we removed the LD when we decomposed the initial matrices
    // from now on, we can treat the buffers as matrices where ld = rows
    f32tof64_flat<<<ceilDiv(A.rows * A.cols, 256U), 256, 0, s0>>>(dA_h, dA_dgemm, A.rows * A.cols);

    f32tof64_flat<<<ceilDiv(B.rows * B.cols, 256U), 256, 0, s2>>>(dB_h, dB_dgemm, B.rows * B.cols);

    // notify that the conversion is done for dB_h
    HANDLE_ERR(hipEventRecord(e0, s2));


    // ###################################################
    // STEP 3: dgemm on dA and dB
    // ###################################################

    // wait for s2 to finish converting dB_h into dB_dgemm
    HANDLE_ERR(hipStreamWaitEvent(s0, e0, 0));

    // perform the dgemm (dRes_dgemm = dA_dgemm * dB_dgemm) on S0
    HANDLE_ERR(hipblasSetStream(handle, s0));

    double beta_main = 0.0;

    HANDLE_ERR(hipblasDgemm(
        handle,
        convertToCublas(transA), convertToCublas(transB),
        m, n, k,
        &alpha,
        dA_dgemm, checked_cast<int>(A.rows),
        dB_dgemm, checked_cast<int>(B.rows),
        &beta_main,
        dRes_dgemm, checked_cast<int>(C.rows)
    ));
    
    // notify that the dgemm on s0 is done
    HANDLE_ERR(hipEventRecord(e0, s0));

    // ###################################################
    // STEP 4: decompose dRes_dgemm into dC_h and dC_l
    // ###################################################

    // perform the decomposition immediatly on s0 because
    // it is the stream which performed the dgemm
    extractf32_mixedhl_flat<<<ceilDiv(C.rows * C.cols, 256U), 256, 0, s0>>>(dRes_dgemm, dC_h, dC_l, C.rows * C.cols);
    
    HANDLE_ERR(hipEventRecord(e0, s0)); // s0 finished the decomposition

    // ###################################################
    // STEP 5: send back dC_h to the host
    // ###################################################

    HANDLE_ERR(hipMemcpyAsync(C_h, dC_h, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost, s0));

    // ###################################################
    // STEP 6: accumulate sgemm rounds
    // ###################################################

    float sgemm_alpha = 1.0f;
    float sgemm_beta = 1.0f;

    // wait for s2 to finish the dC_l decomposition
    // and perform the sgemm dC_l =  dA_h * dB_l + dC_l
    HANDLE_ERR(hipStreamWaitEvent(s1, e0, 0));
    HANDLE_ERR(hipblasSetStream(handle, s1));
    HANDLE_ERR(hipblasSgemm(
        handle,
        convertToCublas(transA), convertToCublas(transB),
        m, n, k,
        &sgemm_alpha,
        dA_l, checked_cast<int>(A.rows),
        dB_h, checked_cast<int>(B.rows),
        &sgemm_beta,
        dC_l, checked_cast<int>(C.rows)
    ));

    // wait for s4 to finish the upload of dC_h
    // and perform the sgemm dC_l = dA_l * dB_h + dC_l
    HANDLE_ERR(hipStreamWaitEvent(s1, e1, 0));
    HANDLE_ERR(hipblasSgemm(
        handle,
        convertToCublas(transA), convertToCublas(transB),
        m, n, k,
        &sgemm_alpha,
        dA_h, checked_cast<int>(A.rows),
        dB_l, checked_cast<int>(B.rows),
        &sgemm_beta,
        dC_l, checked_cast<int>(C.rows)
    ));

    // ###################################################
    // STEP 7: send the result back to the host
    // ###################################################
    
    HANDLE_ERR(hipMemcpyAsync(C_l, dC_l, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost, s1));

    // ###################################################
    // STEP 8 wait for everything to finish
    // ###################################################

    HANDLE_ERR(hipStreamSynchronize(s0));
    HANDLE_ERR(hipStreamSynchronize(s1));
    HANDLE_ERR(hipStreamSynchronize(s3));
    HANDLE_ERR(hipStreamSynchronize(s2));

    perf.compute = std::chrono::high_resolution_clock::now() - start;

    // ###################################################
    // STEP 9: cleanup
    // ###################################################

    HANDLE_ERR(hipStreamDestroy(s1));
    HANDLE_ERR(hipStreamDestroy(s2));
    HANDLE_ERR(hipStreamDestroy(s3));

    HANDLE_ERR(hipEventDestroy(e0));
    HANDLE_ERR(hipEventDestroy(e1));

    // ###################################################
    // STEP 10: precision-recompose C
    // ###################################################

    re_f64f32(C_h, C_l, C.ptr, C.rows, C.cols, C.ld);

    HANDLE_ERR(hipFree(dA_h));
    HANDLE_ERR(hipFree(dA_l));
    HANDLE_ERR(hipFree(dA_dgemm));
    HANDLE_ERR(hipFree(dB_h));
    HANDLE_ERR(hipFree(dB_l));
    HANDLE_ERR(hipFree(dB_dgemm));
    HANDLE_ERR(hipFree(dC_h));
    HANDLE_ERR(hipFree(dC_l));
    HANDLE_ERR(hipFree(dRes_dgemm));

    HANDLE_ERR(hipHostFree(A_h));
    HANDLE_ERR(hipHostFree(A_l));
    HANDLE_ERR(hipHostFree(B_h));
    HANDLE_ERR(hipHostFree(C_h));
    HANDLE_ERR(hipHostFree(C_l));

    HANDLE_ERR(hipDeviceSynchronize());

    return perf;
}

template <typename DataType>
PerfRecord PPMatrix<DataType>::gemm(
    hipblasHandle_t handle,
        char transA,
        char transB,
        DataType alpha,
        const PPMatrix<DataType>& A,
        const PPMatrix<DataType>& B,
        DataType beta,
        PPMatrix<DataType>& C
) {
    assert(A.rows == C.rows);
    assert(B.cols == C.cols);
    assert(A.cols == B.rows);

    DataType *dA, *dB, *dC;

    bool use_beta = !is_literal_zero(beta);

    int m = checked_cast<int>((transA == 'N') ? A.rows : A.cols);
    int n = checked_cast<int>((transB == 'N') ? B.cols : B.rows);
    int k = checked_cast<int>((transA == 'N') ? A.cols : A.rows);

    HANDLE_ERR(hipMalloc(&dA, A.rows * A.cols * sizeof(DataType)));
    HANDLE_ERR(hipMalloc(&dB, B.rows * B.cols * sizeof(DataType)));
    HANDLE_ERR(hipMalloc(&dC, C.rows * C.cols * sizeof(DataType)));

    HANDLE_ERR(hipHostRegister(A.ptr, A.rows * A.cols * sizeof(DataType), hipHostRegisterDefault));
    HANDLE_ERR(hipHostRegister(B.ptr, B.rows * B.cols * sizeof(DataType), hipHostRegisterDefault));

    HANDLE_ERR(hipHostRegister(C.ptr, C.rows * C.cols * sizeof(DataType), hipHostRegisterDefault));

    auto start = std::chrono::high_resolution_clock::now();

    HANDLE_ERR(hipMemcpy(dA, A.ptr, A.rows * A.cols * sizeof(DataType), hipMemcpyHostToDevice));
    HANDLE_ERR(hipMemcpy(dB, B.ptr, B.rows * B.cols * sizeof(DataType), hipMemcpyHostToDevice));

    if (use_beta) {
        HANDLE_ERR(hipMemcpy(dC, C.ptr, C.rows * C.cols * sizeof(DataType), hipMemcpyHostToDevice));
    }

    hipDeviceSynchronize();

    auto h2dDone = std::chrono::high_resolution_clock::now();

    if constexpr (std::is_same_v<DataType, f32>) {
        HANDLE_ERR(hipblasSgemm(
            handle,
            convertToCublas(transA), convertToCublas(transB),
            m, n, k,
            &alpha,
            dA, checked_cast<int>(A.rows),
            dB, checked_cast<int>(B.rows),
            &beta,
            dC, checked_cast<int>(C.rows)
        ));
    } else {
        static_assert(std::is_same_v<DataType, f64>, "Unsupported data type (only f32 and f64 are supported).");
        HANDLE_ERR(hipblasDgemm(
            handle,
            convertToCublas(transA), convertToCublas(transB),
            m, n, k,
            &alpha,
            dA, checked_cast<int>(A.rows),
            dB, checked_cast<int>(B.rows),
            &beta,
            dC, checked_cast<int>(C.rows)
        ));
    }

    hipDeviceSynchronize();

    auto computeDone = std::chrono::high_resolution_clock::now();

    HANDLE_ERR(hipMemcpy(C.ptr, dC, C.rows * C.cols * sizeof(DataType), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    auto d2hDone = std::chrono::high_resolution_clock::now();

    HANDLE_ERR(hipFree(dA));
    HANDLE_ERR(hipFree(dB));
    HANDLE_ERR(hipFree(dC));

    HANDLE_ERR(hipHostUnregister(A.ptr));
    HANDLE_ERR(hipHostUnregister(B.ptr));

    if (use_beta) {
        HANDLE_ERR(hipHostUnregister(C.ptr));
    }

    return PerfRecord{ h2dDone - start, computeDone - h2dDone, d2hDone - computeDone };
}

template <typename DataType>
void PPMatrix<DataType>::blasGemm(
    char transA,
    char transB,
    DataType alpha,
    const PPMatrix<DataType>& A,
    const PPMatrix<DataType>& B,
    DataType beta,
    PPMatrix<DataType>& C
) {
    assert(A.rows == C.rows);
    assert(B.cols == C.cols);
    assert(A.cols == B.rows);

    int m = checked_cast<int>((transA == 'N') ? A.rows : A.cols);
    int n = checked_cast<int>((transB == 'N') ? B.cols : B.rows);
    int k = checked_cast<int>((transA == 'N') ? A.cols : A.rows);

    if constexpr (std::is_same_v<DataType, f32>) {
        sgemm_(&transA, &transB, &m, &n, &k, &alpha, A.ptr, &m, B.ptr, &k, &beta, C.ptr, &m);
    } else {
        static_assert(std::is_same_v<DataType, f64>, "Unsupported data type (only f32 and f64 are supported).");
        dgemm_(&transA, &transB, &m, &n, &k, &alpha, A.ptr, &m, B.ptr, &k, &beta, C.ptr, &m);
    }
}

template <typename DataType>
void PPMatrix<DataType>::sub(
    const PPMatrix<DataType>& A,
    const PPMatrix<DataType>& B,
    PPMatrix<DataType>& C
) {
    assert((A.rows == B.rows) && (A.rows == B.rows));
    assert((A.cols == B.cols) && (A.cols == B.cols));

    for (u32 i = 0; i < A.rows; ++i) {
        for (u32 j = 0; j < A.cols; ++j) {
            C.ptr[i * C.ld + j] = A.ptr[i * A.ld + j] - B.ptr[i * B.ld + j];
        }
    }
}

template <typename DataType>
DataType PPMatrix<DataType>::norm(char norm) const {
    int M   = checked_cast<int>(this->rows);
    int N   = checked_cast<int>(this->cols);
    int LD  = checked_cast<int>(this->ld);

    if constexpr (std::is_same_v<DataType, f32>) {
        return slange_(&norm, &M, &N, this->ptr, &LD, nullptr);
    } else {
        static_assert(std::is_same_v<DataType, f64>, "Unsupported data type (only f32 and f64 are supported).");
        return dlange_(&norm, &M, &N, this->ptr, &LD, nullptr);
    }
}


template <typename DataType>
DataType PPMatrix<DataType>::norm2() const {
    char JOBU = 'N';
    char JOBVT = 'N';
    int M   = checked_cast<int>(this->rows);
    int N   = checked_cast<int>(this->cols);
    int LDA = checked_cast<int>(this->ld);

    auto lw = std::min(this->rows, this->cols);
    auto lwork_base = std::min(this->rows, this->cols) * 50;

    auto S = std::vector<DataType>(lw);
    int LDU = 1;
    int LDVT = 1;
    int LWORK = checked_cast<int>(lwork_base);
    auto WORK = std::vector<DataType>(lwork_base);
    int INFO = 0;

    if constexpr (std::is_same_v<DataType, f32>) {
        sgesvd_(&JOBU, &JOBVT, &M, &N, this->ptr, &LDA, S.data(), nullptr, &LDU, nullptr, &LDVT, WORK.data(), &LWORK, &INFO);
    } else {
        static_assert(std::is_same_v<DataType, f64>, "Unsupported data type (only f32 and f64 are supported).");
        dgesvd_(&JOBU, &JOBVT, &M, &N, this->ptr, &LDA, S.data(), nullptr, &LDU, nullptr, &LDVT, WORK.data(), &LWORK, &INFO);
    }

    return S[0];
}

template class PPMatrix<f32>;
template class PPMatrix<f64>;
