#include <stdexcept>

#include "ppmatrix.hpp"
#include "pputils.hpp"
#include "../util/helper.hpp"

#include "fmt/core.h"


__host__ void ppgemm_f32(
    hipblasHandle_t handle,
    char transA,
    char transB,
    f32 alpha,
    PPMatrix<f32>& A,
    PPMatrix<f32>& B,
    f32 beta,
    PPMatrix<f32>& C
) {
    fmt::print("/!\\ not implemented\n");
    throw std::exception();
}
 
static void handle_err(hipError_t val, int line) {
    if (__builtin_expect(val != hipSuccess, 0)) {
        fmt::print("CUDA error at line {}: {}\n", line, hipGetErrorString(hipGetLastError()));
        throw std::exception();
    }
}

__host__ void ppgemm_f64(
    hipblasHandle_t handle,
    char transA,
    char transB,
    f64 alpha,
    PPMatrix<f64>& A,
    PPMatrix<f64>& B,
    f64 beta,
    PPMatrix<f64>& C
) {
    assert(A.rows == C.rows);
    assert(B.cols == C.cols);
    assert(A.cols == B.rows);

    int m = static_cast<int>((transA == 'N') ? A.rows : A.cols);
    int n = static_cast<int>((transB == 'N') ? B.cols : B.rows);
    int k = static_cast<int>((transA == 'N') ? A.cols : A.rows);

    // ####################
    // STEP 0: preparation
    // ####################

    // alloc CPU buffers
    float *A_h, *A_l, *B_h, *B_l, *C_h, *C_l;
    
    handle_err(hipHostMalloc(&A_h, A.rows * A.cols * sizeof(float)), __LINE__);
    handle_err(hipHostMalloc(&A_l, A.rows * A.cols * sizeof(float)), __LINE__);

    handle_err(hipHostMalloc(&B_h, B.rows * B.cols * sizeof(float)), __LINE__);
    handle_err(hipHostMalloc(&B_l, B.rows * B.cols * sizeof(float)), __LINE__);

    handle_err(hipHostMalloc(&C_h, C.rows * C.cols * sizeof(float)), __LINE__);
    handle_err(hipHostMalloc(&C_l, C.rows * C.cols * sizeof(float)), __LINE__);

    // alloc GPU buffers
    float *dA_h, *dA_l, *dB_h, *dB_l, *dC_h, *dC_l;
    double *dA_dgemm, *dB_dgemm, *dRes_dgemm;

    handle_err(hipMalloc(&dA_h, A.rows * A.cols * sizeof(float)), __LINE__);
    handle_err(hipMalloc(&dA_l, A.rows * A.cols * sizeof(float)), __LINE__);

    handle_err(hipMalloc(&dB_h, B.rows * B.cols * sizeof(float)), __LINE__);
    handle_err(hipMalloc(&dB_l, B.rows * B.cols * sizeof(float)), __LINE__);

    handle_err(hipMalloc(&dC_h, C.rows * C.cols * sizeof(float)), __LINE__);
    handle_err(hipMalloc(&dC_l, C.rows * C.cols * sizeof(float)), __LINE__);

    handle_err(hipMalloc(&dA_dgemm, A.rows * A.cols * sizeof(double)), __LINE__);
    handle_err(hipMalloc(&dB_dgemm, B.rows * B.cols * sizeof(double)), __LINE__);
    handle_err(hipMalloc(&dRes_dgemm, C.rows * C.cols * sizeof(double)), __LINE__);

    // use the default stream so that no sync are needed before / after a call to this function
    hipStream_t s0 = 0;

    // register streams
    // streams are created as hipStreamNonBlocking, meaning they do not
    // implicitly sync with the default stream
    hipStream_t s1, s2, s3;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s3, hipStreamNonBlocking);

    // register events
    hipEvent_t e0, e1;
    hipEventCreate(&e0);
    hipEventCreate(&e1);

    // ###########################################
    // STEP 1: precision-decompose and send A & B
    // ###########################################

    // No need to use events here, communications from/to the device are serialized

    // decompose A on the host
    de_f64f32(A.ptr, A_h, A_l, A.rows, A.cols, A.ld);

    
    // copy A_h to GPU
    hipMemcpyAsync(dA_h, A_h, A.rows * A.cols * sizeof(float), hipMemcpyHostToDevice, s0);

    // decompose B on the host (we're already copying A_h to GPU, so we can do this in parallel)
    de_f64f32(B.ptr, B_h, B_l, B.rows, B.cols, B.ld);

    // copy B_h to GPU
    hipMemcpyAsync(dB_h, B_h, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice, s2);

    // copy A_l to GPU
    hipMemcpyAsync(dA_l, A_l, A.rows * A.cols * sizeof(float), hipMemcpyHostToDevice, s1);

    // copy B_l to GPU
    hipMemcpyAsync(dB_l, B_l, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice, s3);
    hipEventRecord(e1, s3);

    // ###################################################
    // STEP 2: convert dA_h and dB_h to double
    // ###################################################

    // launch the kernel in blocks of 256 threads
    // we removed the LD when we decomposed the initial matrices
    // from now on, we can treat the buffers as matrices where ld = rows
    f32tof64_flat<<<ceilDiv(A.rows * A.cols, 256U), 256, 0, s0>>>(dA_h, dA_dgemm, A.rows * A.cols);

    f32tof64_flat<<<ceilDiv(B.rows * B.cols, 256U), 256, 0, s2>>>(dB_h, dB_dgemm, B.rows * B.cols);

    // notify that the conversion is done for dB_h
    hipEventRecord(e0, s2);


    // ###################################################
    // STEP 3: dgemm on dA and dB
    // ###################################################

    // wait for s2 to finish converting dB_h into dB_dgemm
    hipStreamWaitEvent(s0, e0, 0);

    // perform the dgemm (dRes_dgemm = dA_dgemm * dB_dgemm) on S0
    hipblasSetStream(handle, s0);

    hipblasDgemm(
        handle,
        convertToCublas(transA), convertToCublas(transB),
        m, n, k,
        &alpha,
        dA_dgemm, A.rows,
        dB_dgemm, B.rows,
        &beta,
        dRes_dgemm, C.rows
    );
    
    // notify that the dgemm on s0 is done
    hipEventRecord(e0, s0);

    // ###################################################
    // STEP 4: decompose dRes_dgemm into dC_h and dC_l
    // ###################################################

    // perform the decomposition immediatly on s0 because
    // it is the stream which performed the dgemm
    extractf32high_flat<<<ceilDiv(C.rows * C.cols, 256U), 256, 0, s0>>>(dRes_dgemm, dC_h, C.rows * C.cols);

    // wait for s0 to finish dgemm and perform the decomposition on s2
    hipStreamWaitEvent(s2, e0, 0);
    extractf32low_flat<<<ceilDiv(C.rows * C.cols, 256U), 256, 0, s2>>>(dRes_dgemm, dC_l, C.rows * C.cols);
    hipEventRecord(e0, s2); // s2 finished the decomposition

    // ###################################################
    // STEP 5: send back dC_h to the host
    // ###################################################

    hipMemcpyAsync(C_h, dC_h, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost, s0);

    // ###################################################
    // STEP 6: accumulate sgemm rounds
    // ###################################################

    float sgemm_alpha = 1.0f;
    float sgemm_beta = 1.0f;

    // wait for s2 to finish the dC_l decomposition
    // and perform the sgemm dC_l =  dA_h * dB_l + dC_l
    hipStreamWaitEvent(s1, e0, 0);
    hipblasSetStream(handle, s1);
    hipblasSgemm(
        handle,
        convertToCublas(transA), convertToCublas(transB),
        m, n, k,
        &sgemm_alpha,
        dA_h, A.rows,
        dB_l, B.rows,
        &sgemm_beta,
        dC_l, C.rows
    );

    // wait for s4 to finish the upload of dC_h
    // and perform the sgemm dC_l = dA_l * dB_h + dC_l
    hipStreamWaitEvent(s1, e1, 0);
    hipblasSgemm(
        handle,
        convertToCublas(transA), convertToCublas(transB),
        m, n, k,
        &sgemm_alpha,
        dA_l, A.rows,
        dB_h, B.rows,
        &sgemm_beta,
        dC_l, C.rows
    );

    // ###################################################
    // STEP 7: send the result back to the host
    // ###################################################
    
    hipMemcpyAsync(C_l, dC_l, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost, s1);

    // ###################################################
    // STEP 8 wait for everything to finish
    // ###################################################

    hipStreamSynchronize(s0);
    hipStreamSynchronize(s1);
    hipStreamSynchronize(s3);
    hipStreamSynchronize(s2);

    // ###################################################
    // STEP 9: cleanup
    // ###################################################

    hipStreamDestroy(s0);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);

    hipEventDestroy(e0);
    hipEventDestroy(e1);

    // ###################################################
    // STEP 10: precision-recompose C
    // ###################################################
    re_f64f32(C_h, C_l, C.ptr, C.rows, C.cols, C.ld);

    hipFree(dA_h);
    hipFree(dA_l);
    hipFree(dA_dgemm);
    hipFree(dB_h);
    hipFree(dB_l);
    hipFree(dB_dgemm);
    hipFree(dC_h);
    hipFree(dC_l);
    hipFree(dRes_dgemm);

    hipHostFree(A_h);
    hipHostFree(A_l);
    hipHostFree(B_h);
    hipHostFree(C_h);
    hipHostFree(C_l);
}
